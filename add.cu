
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate memory on host
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array on host
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }
    

    //allocate device memory
    hipMalloc((void**)&d_a, sizeof(float)*N);
    hipMalloc((void**)&d_b, sizeof(float)*N);
    hipMalloc((void**)&d_out, sizeof(float)*N);

    //transfer arrays a and b from host to device
    hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);

    // Main function
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);
    
    //transfer out data from device to host
    hipMemcpy(out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++){
        printf("here's out[i]: %f \n", (out[i]));
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);
}