/*
hello.cu is a cuda file
.cu for cuda
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void hello(){
    printf("I am so cool! I can code GPU! \n");
}


int main(){
    hello<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}